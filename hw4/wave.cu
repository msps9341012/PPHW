/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void __global__ update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 

   /* Initialize old values array */
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update(float *values_gpu,int tpoints,int nsteps)
{
   int threadID;
   threadID=blockIdx.x*blockDim.x+threadIdx.x;
   
   if(threadID<=tpoints){
	
	float values_t;
	float new_val;
	float old_val;

	float x,fac,tmp;
	fac=2.0*PI;
	tmp=tpoints-1;
	x=(float)(threadID-1)/tmp;
	values_t=sin(fac*x);
	old_val=values_t;

	for(int i=1;i<=nsteps;i++){
		if((threadID==1)||(threadID==tpoints))
			new_val=0.0;
		else
			new_val=(2.0*values_t)-old_val+(0.09*(-2.0*values_t));

		old_val=values_t;
		values_t=new_val;
	}
	
	values_gpu[threadID]=values_t;


   }	   

}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	float *values_gpu;
	int size;

	check_param();
	printf("Initializing points on the line...\n");
	//init_line();
	printf("Updating all points for all time steps...\n");
	
	int count;
	int maxThreadsPerBlock;
	hipGetDeviceCount(&count);
	int i;
	for(i=0;i<count;i++){
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop,i)== hipSuccess){
			maxThreadsPerBlock=prop.maxThreadsPerBlock;
		}
	}	
	hipSetDevice(i);

	size=(1+tpoints)*sizeof(float);
	
	hipMalloc((void**)&values_gpu,size);
	update<<<(tpoints/maxThreadsPerBlock),maxThreadsPerBlock>>>(values_gpu,tpoints,nsteps);
	hipMemcpy(values,values_gpu,size,hipMemcpyDeviceToHost);	

	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}
